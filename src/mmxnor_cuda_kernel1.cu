#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "mmxnor_cuda_kernel.h"


//horizontal
__global__ void AMatrix2Bin(float *a,int *a_bin,int a_rows,int pitch_a,int pitch_a_bin,int MaxBS,int BINSIZE){
    int tix=threadIdx.x;
    // int tiy=threadIdx.y;
    int bix=blockIdx.x;
    // int biy=blockIdx.y;
    int bdx=blockDim.x;
    // int bdy=blockDim.y;
    int gdx=gridDim.x;
    // int gdy=gridDim.y;


    int maxThreads=MaxBS*a_rows;
    for(int id = bix*bdx+tix; id < maxThreads; id+=gdx*bdx) {
        int rid=id/MaxBS;
        int cid=id%MaxBS;

        int Integer=0;
        int base=1;
        for (int i=0;i<BINSIZE;i++){
            if (a[rid*pitch_a+(cid+1)*BINSIZE-1-i]==1.f){
                Integer+=base;
            }
            base=base<<1;
        }

        a_bin[rid*pitch_a_bin+cid]=Integer;
    }

}
//vetical
__global__ void BMatrix2Bin(float *b,int *b_bin,int b_cols,int pitch_b,int pitch_b_bin,int MaxBS,int BINSIZE){
    int tix=threadIdx.x;
    // int tiy=threadIdx.y;
    int bix=blockIdx.x;
    // int biy=blockIdx.y;
    int bdx=blockDim.x;
    // int bdy=blockDim.y;
    int gdx=gridDim.x;
    // int gdy=gridDim.y;

    int maxThreads=MaxBS*b_cols;
    for(int id = bix*bdx+tix; id < maxThreads; id+=gdx*bdx) {
        int cid=id/MaxBS;
        int rid=id%MaxBS;

        int Integer=0;
        int base=1;
        for (int i=0;i<BINSIZE;i++){
            if (b[((rid+1)*BINSIZE-1-i)*pitch_b+cid]==1.f){
                Integer+=base;
            }
            base=base<<1;
        }

        b_bin[rid*pitch_b_bin+cid]=Integer;
    }

}

__device__ unsigned char __popcount_tab_device[256];//__constant__ is slower than __device__
__device__ int popcount (int x) {
  return __popcount_tab_device[(x >>  0) & 0xff]  
  + __popcount_tab_device[(x >>  8) & 0xff]  
  + __popcount_tab_device[(x >> 16) & 0xff] 
  + __popcount_tab_device[(x >> 24) & 0xff];
}

__global__ void MatrixMulXnor(int *a,int *b,int a_rows,int a_cols,
    int b_cols,float *result,int pitch_a,int pitch_b,
    int pitch_result,int BINSIZE,int RealMidSize){

    int tix=threadIdx.x;
    // int tiy=threadIdx.y;
    int bix=blockIdx.x;
    // int biy=blockIdx.y;
    int bdx=blockDim.x;
    // int bdy=blockDim.y;
    int gdx=gridDim.x;
    // int gdy=gridDim.y;

    int rest=(BINSIZE*a_cols-RealMidSize);
    for(int i=bix;i<a_rows;i+=gdx){
        for(int j=tix;j<b_cols;j+=bdx){
            // printf("i=%d ; j=%d\n",i,j);
            int sum=0;
            for(int k=0;k<a_cols;k++){
                int bin=(a[i*pitch_a+k]^b[k*pitch_b+j]);
                int negnum=popcount(bin);
                int posnum=BINSIZE-negnum;
                //calculate ignores the rest of BINSIZE if the Matsize cant devided by BINSIZE ,it can cause err
                //(10/00)'(01/00) should be 0000 but it is 0011,so 1+1 is trash in the result.and it can cause a_rows*b_cols times. 
                sum+=(posnum-negnum);
            }
            result[i*pitch_result+j]=sum-rest;
        }
    }


}


void MatrixMul_Xnor(float *a,float *b,int a_rows,int a_cols,int b_cols,float *result,hipStream_t stream){

    int BINSIZE=30;//size of bin2int, 32 means 0000 0000 0000 0000 0000 0000 0000 0000
    int MaxBS=(a_cols-1)/BINSIZE+1;
    int a_cols_Copysize=MaxBS*BINSIZE;
    dim3 BS_BIN(512,1,1);
    dim3 GS_BIN(6,1,1);
    // fprintf(stderr, "MaxBS : %d\n", MaxBS);

    float *a_device;//a_rows * a_cols_Copysize
    float *b_device;//a_cols_Copysize * b_cols
    size_t pitch_a_device, pitch_b_device;
    hipMallocPitch((void**)&a_device , &pitch_a_device , sizeof(float) *a_cols_Copysize , a_rows);
    hipMallocPitch((void**)&b_device , &pitch_b_device , sizeof(float) *b_cols , a_cols_Copysize);
    hipMemset(a_device, 0, pitch_a_device * a_rows);
    hipMemset(b_device, 0, pitch_b_device * a_cols_Copysize);
    hipMemcpy2D(a_device,pitch_a_device,a,sizeof(float) *a_cols ,sizeof(float) *a_cols, a_rows,hipMemcpyDeviceToDevice);
    hipMemcpy2D(b_device,pitch_b_device,b,sizeof(float) *b_cols ,sizeof(float) *b_cols, a_cols,hipMemcpyDeviceToDevice);


    int *a_device_bin;
    int *b_device_bin;
    size_t pitch_a_device_bin, pitch_b_device_bin;
    hipMallocPitch((void**)&a_device_bin , &pitch_a_device_bin , sizeof(int) *MaxBS , a_rows);
    hipMallocPitch((void**)&b_device_bin , &pitch_b_device_bin , sizeof(int) *b_cols , MaxBS);

    AMatrix2Bin<<<GS_BIN,BS_BIN>>>(a_device , a_device_bin , a_rows , 
        pitch_a_device/sizeof(float) , pitch_a_device_bin/sizeof(int) , MaxBS , BINSIZE);
    BMatrix2Bin<<<GS_BIN,BS_BIN>>>(b_device , b_device_bin , b_cols , 
        pitch_b_device/sizeof(float) , pitch_b_device_bin/sizeof(int) , MaxBS , BINSIZE);


    float *result_device;//a_rows * b_cols
    size_t pitch_result_device;
    hipMallocPitch((void**)&result_device , &pitch_result_device , sizeof(float) *b_cols , a_rows);

    const unsigned char __popcount_tab[] = {
      0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8,
    };
    hipMemcpyToSymbol(HIP_SYMBOL(__popcount_tab_device), __popcount_tab, sizeof(__popcount_tab));


    dim3 BS_MM(1024,1,1);
    dim3 GS_MM(120,1,1);
    MatrixMulXnor<<<GS_MM,BS_MM>>>(a_device_bin , b_device_bin , a_rows , MaxBS , b_cols ,
     result_device , pitch_a_device_bin/sizeof(int) , pitch_b_device_bin/sizeof(int) , 
     pitch_result_device/sizeof(float) , BINSIZE , a_cols);


    hipMemcpy2D(result,sizeof(float) *b_cols, result_device,pitch_result_device,sizeof(float) *b_cols , a_rows ,hipMemcpyDeviceToDevice);

    hipFree(a_device);
    hipFree(b_device);
    hipFree(a_device_bin);
    hipFree(b_device_bin);
    hipFree(result_device);


    hipError_t err;
    err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}